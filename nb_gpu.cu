
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <map>
#include <string>
#include <sstream>
#include <iterator>
#include <algorithm>
#include <hip/hip_runtime_api.h>
/*
 * Fills in the matrix term_class_matrix based on the frequency of terms. The term_index_arr
 * holds the indices for the doc_term_arr where each term starts. Increment frequency of
 * term_class_matrix for the class and term by looping through all docs with that term.
 * The doc_class array is used to hold the class of each doc
 */
__global__ void calcFreq(int *term_index_arr, int *doc_term_arr, int *doc_class, float *term_class_matrix,
							int num_terms, int doc_term_len, int classes) {
	unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	int start = term_index_arr[i];
	int end = term_index_arr[i];
	if(i < num_terms - 1) {
		end = term_index_arr[i+1];
	} else if (i == num_terms - 1){
		end = doc_term_len - 1;
	} else {
		return ;
	}

	for (int x = start; x < end; x++) {
		term_class_matrix[classes * i + doc_class[doc_term_arr[x] - 1]] += 1.0;
	}
}

/*
 * Calculates total number of terms per class and places into an array. Parallelized
 * based on class
 */
__global__ void calcTotalTermsPerClass(float * term_class_matrix, int * terms_per_class, int num_terms, int classes) {
	unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	if (i < classes) {
		int sum = 0;
		for (int x = 0; x < num_terms; x++) {
			sum += term_class_matrix[classes * x + i];
		}
		terms_per_class[i] = sum;
	}
}


/*
 * Goes through each term and divides the term frequency in the class by the total
 * terms in that class. Parallelized based on terms
 */
__global__ void learn(float * term_class_matrix, int num_docs, int classes, int * terms_per_class, int num_terms) {
	unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	if (i < num_terms) {
		for (int x = 0; x < classes; x++) {
			term_class_matrix[classes * i + x] /= terms_per_class[x];
		}
	}
}

__global__ void test(float *term_class_matrix, float * doc_prob, int * doc_index, int * terms_in_doc, int classes, int num_docs, int total_len_terms, int *predictions) {
	unsigned int i = blockIdx.x * gridDim.y * gridDim.z *
                      blockDim.x + blockIdx.y * gridDim.z *
                      blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	int start_term = doc_index[i];
	int end_term = doc_index[i];
	if(i < num_docs - 1) {
		end_term = doc_index[i+1];
	} else if (i == num_docs - 1) {
		end_term = total_len_terms - 1;
	} else {
		return ;
	}
	for (int x = start_term; x < end_term; x++) {
		for (int y = 0; y < classes; y++) {
			doc_prob[classes * i + y] += log(term_class_matrix[classes * x + y]);
		}
	}
	int max_index = 0;
	float max = -3.40282e+038;
	for (int y = 0; y < classes; y++) {
		if (doc_prob[classes * i + y] > max) {
			max_index = y;
		}
	}
	predictions[i] = max_index;

}

void errorCheck(hipError_t err) {
	if (err) {
		fprintf(stderr, "CUDA error: %d\n", err);
		exit(err);
	}
}

static hipError_t numBlocksThreads(unsigned int N, dim3 *numBlocks, dim3 *threadsPerBlock) {
    unsigned int BLOCKSIZE = 128;
    int Nx, Ny, Nz;
    int device;
    hipError_t err;
    if(N < BLOCKSIZE) {
        numBlocks->x = 1;
        numBlocks->y = 1;
        numBlocks->z = 1;
        threadsPerBlock->x = N;
        threadsPerBlock->y = 1;
        threadsPerBlock->z = 1;
        return hipSuccess;
    }
    threadsPerBlock->x = BLOCKSIZE;
    threadsPerBlock->y = 1;
    threadsPerBlock->z = 1;
    err = hipGetDevice(&device);
    if(err)
      return err;
    err = hipDeviceGetAttribute(&Nx, hipDeviceAttributeMaxBlockDimX, device);
    if(err)
      return err;
    err = hipDeviceGetAttribute(&Ny, hipDeviceAttributeMaxBlockDimY, device);
    if(err)
      return err;
    err = hipDeviceGetAttribute(&Nz, hipDeviceAttributeMaxBlockDimZ, device);
    if(err)
      return err;
    unsigned int n = (N-1) / BLOCKSIZE + 1;
    unsigned int x = (n-1) / (Ny*Nz) + 1;
    unsigned int y = (n-1) / (x*Nz) + 1;
    unsigned int z = (n-1) / (x*y) + 1;
    if(x > Nx || y > Ny || z > Nz) {
        return hipErrorInvalidConfiguration;
    }
    numBlocks->x = x;
    numBlocks->y = y;
    numBlocks->z = z;

    return hipSuccess;
}

/* Function to convert vector of ints into array of ints */
int * vecToArr(std::vector<int> v)
{
	int *arr = (int *)malloc(v.size() * sizeof(int));
	if(arr == NULL)
	{
		std::cerr << "Error converting vector to array" << std::endl;
		exit(-1);
	}

	std::copy(v.begin(), v.end(), arr);

	return arr;
}


int main(int argc, char **argv)
{
	if(argc != 3)
	{
		std::cerr << "Usage: " << argv[0] << " [train_file] [test_file]" << std::endl;
		exit(-1);
	}

	/* Use vector to store terms */
	std::vector<std::string> term_vec;
	int term_index = 0;

	/* Map of term to document list, to make sure no duplicate documents are added to list */
	std::map<std::string, std::vector<int> > term_doc_map;

	/*
		Vector of terms.
		Each index represents the term.
		The value at that index represents the index in doc_term that holds list of documents for the term
		Note: Will be converted to array later (to be used in kernel function)
	*/
	std::vector<int> term_index_vec;


	/*
		Vector of documents.
		Each value represents the doc_number that the term has appeared in
		Note: Will be converted to array later (to be used in kernel function)
	*/
	std::vector<int> doc_term_vec;

	std::vector<int> doc_class;

	/* Vector to hold all the classes */
	std::vector<std::string> classes_vec;

	/* Loop through each document */
	std::ifstream file(argv[1]);
	std::string line;
	int lineno = 0;
	while (std::getline(file, line))
	{
		/*
			Split string
			doc_split[0] = doc_class
			doc_split[1 -> end] = terms in doc
		*/
		std::istringstream iss(line);
		std::vector<std::string> doc_split((std::istream_iterator<std::string>(iss)),
                                 std::istream_iterator<std::string>());
		//doc_split.push_back(std::to_string(lineno));

		/* Append class to classes_vec, only if it has not been seen before */
		std::vector<std::string>::iterator class_it = std::find(classes_vec.begin(), classes_vec.end(), doc_split[0]);
        if(class_it == classes_vec.end())
            classes_vec.push_back(doc_split[0]);

		doc_class.push_back(find(classes_vec.begin(), classes_vec.end(), doc_split[0]) - classes_vec.begin());

        /* Loop through each term in the document */
        for(int i = 1; i < doc_split.size(); i++)
        {
        	std::string term = doc_split[i];

        	/* Add term to vector list, if not done so already */
        	std::vector<std::string>::iterator term_it = std::find(term_vec.begin(), term_vec.end(), term);
        	if(term_it == term_vec.end())
            	term_vec.push_back(term);

        	/* Add the document to the list of documents for this term, if not done so already */
        	std::vector<int> doc_list = term_doc_map[term];
        	std::vector<int>::iterator doc_it = std::find(doc_list.begin(), doc_list.end(), lineno);
        	if(doc_it == doc_list.end()) {
				doc_list.push_back(lineno);
				term_doc_map[term] = doc_list;
			}
        }
		lineno++;
	}

	/* Go through each term and populate the term_index_vec and doc_term_vec */
	for(int idx = 0; idx < term_vec.size(); idx++)
	{
		/* t is the term itself, idx is its index (in term_index_vec as well) */
		std::string t = term_vec[idx];

		/* d is the list of docs associated with t */
		std::vector<int> d = term_doc_map[t];

		/* The starting index for the list of docs (related to t) is the size of the doc_term_vec before we insert the new docs */
		term_index_vec.push_back(doc_term_vec.size());

		/* Insert the related documents in the doc_term_vec */
		doc_term_vec.insert(doc_term_vec.end(), d.begin(), d.end());

	}

	std::ifstream test_file(argv[2]);
	/*
		Vector of Test documents.
		Each index represents a test document.
		The value at that index represents the index in test_term_doc_vec that holds list of terms for that documents
		Note: Will be converted to array later (to be used in kernel function)
	*/
	std::vector<int> test_doc_index_vec;

	/*
		Vector of valid test document terms.
		Each value represents the term_number that is valid and appears in the document
		Note: Will be converted to array later (to be used in kernel function)
	*/
	std::vector<int> test_term_doc_vec;

	while (std::getline(test_file, line)) {
		std::istringstream iss(line);
		std::vector<std::string> doc_split((std::istream_iterator<std::string>(iss)), std::istream_iterator<std::string>());

		std::vector<int> test_doc_terms;
		for(int i = 0; i < doc_split.size(); i++) {
			std::string term = doc_split[i];
			std::vector<std::string>::iterator term_it = std::find(term_vec.begin(), term_vec.end(), term);
			if (term_it != term_vec.end()) {
				test_doc_terms.push_back(term_it - term_vec.begin());
			} else {
				continue;
			}
		}

		test_doc_index_vec.push_back(test_term_doc_vec.size());
		test_term_doc_vec.insert(test_term_doc_vec.end(), test_doc_terms.begin(), test_doc_terms.end());
	}

	/* Convert the vectors to arrays for GPU processing */
	int *term_index_arr = vecToArr(term_index_vec);
	int *doc_term_arr = vecToArr(doc_term_vec);
	int *doc_class_arr = vecToArr(doc_class);

	/* Create a TxC matrix (i.e. # of Terms x # of Classes) which will hold the frequencies of each term */
	float *term_class_matrix = (float *)calloc( (term_vec.size()) * (classes_vec.size()), sizeof(float) );

	/* Create a C length array holding the total terms in each class*/
	int *total_terms_class_arr = (int *)calloc( classes_vec.size(), sizeof(int));

	size_t nSpatial;
	size_t mSpatial;
	dim3 spatialThreadsPerBlock, spatialBlocks;

	float *d_term_class;
	int *d_term_index;
	int *d_doc_term;
	int *d_doc_class;
	int *d_total_terms_class;

	hipDeviceReset();
    hipProfilerStart();


	nSpatial = doc_term_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_doc_term, mSpatial));
	errorCheck(hipMemcpy(d_doc_term, doc_term_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	nSpatial = doc_class.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_doc_class, mSpatial));
	errorCheck(hipMemcpy(d_doc_class, doc_class_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	nSpatial = term_vec.size() * classes_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
	errorCheck(hipMalloc(&d_term_class, mSpatial));
	errorCheck(hipMemcpy(d_term_class, term_class_matrix, nSpatial*sizeof(float), hipMemcpyHostToDevice));

	nSpatial = term_index_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_term_index, mSpatial));
	errorCheck(hipMemcpy(d_term_index, term_index_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	// Learn
	calcFreq<<<spatialBlocks, spatialThreadsPerBlock>>>(d_term_index, d_doc_term, d_doc_class, d_term_class, term_vec.size(), doc_term_vec.size(), classes_vec.size());

	nSpatial = classes_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_total_terms_class, mSpatial));
	errorCheck(hipMemcpy(d_total_terms_class, total_terms_class_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	calcTotalTermsPerClass<<<spatialBlocks, spatialThreadsPerBlock>>>(d_term_class, d_total_terms_class, term_vec.size(), classes_vec.size());

	nSpatial = term_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	learn<<<spatialBlocks, spatialThreadsPerBlock>>>(d_term_class, doc_class.size(), classes_vec.size(), d_total_terms_class, term_vec.size());

	// Test
	int *test_doc_index_arr = vecToArr(test_doc_index_vec);
	int *test_term_doc_arr = vecToArr(test_term_doc_vec);

	int *predictions = (int *) calloc(test_doc_index_vec.size(), sizeof(int));

	float *test_doc_prob = (float *)calloc( (test_doc_index_vec.size()) * (classes_vec.size()), sizeof(float) );
	float *d_test_doc_prob;

	int *d_test_doc_index;
	int *d_test_term_doc;
	int *d_predictions;

	nSpatial = test_doc_index_vec.size() * classes_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
	errorCheck(hipMalloc(&d_test_doc_prob, mSpatial));
	errorCheck(hipMemcpy(d_test_doc_prob, test_doc_prob, nSpatial*sizeof(float), hipMemcpyHostToDevice));

	nSpatial = test_term_doc_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_test_term_doc, mSpatial));
	errorCheck(hipMemcpy(d_test_term_doc, test_term_doc_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	nSpatial = test_doc_index_vec.size();
	errorCheck(numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock));
	mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(int);
	errorCheck(hipMalloc(&d_test_doc_index, mSpatial));
	errorCheck(hipMemcpy(d_test_doc_index, test_doc_index_arr, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	errorCheck(hipMalloc(&d_predictions, mSpatial));
	errorCheck(hipMemcpy(d_predictions, predictions, nSpatial*sizeof(int), hipMemcpyHostToDevice));

	test<<<spatialBlocks, spatialThreadsPerBlock>>>(d_term_class, d_test_doc_prob, d_test_doc_index, d_test_term_doc, classes_vec.size(), test_doc_index_vec.size(), test_term_doc_vec.size(), d_predictions);

	errorCheck(hipMemcpy(predictions, d_predictions, nSpatial*sizeof(int), hipMemcpyDeviceToHost));
	std::cerr << "Size of predictions: " << sizeof(predictions)/sizeof(int) << std::endl;
	std::cerr << "Size of tests: " << test_doc_index_vec.size() << std::endl;
	std::ofstream results("./results.txt");
	if(results.is_open()) {
		for (int i = 0; i < test_doc_index_vec.size(); i++) {
			results << classes_vec[predictions[i]] << '\n';
		}
	}

	/* Testing stuff */
	// std::cout << "There are " << term_vec.size() << " terms." << std::endl;
	// std::cout << "There are " << lineno << " docs." << std::endl;
	// std::cout << "There are " << classes_vec.size() << " classes." << std::endl;


}
